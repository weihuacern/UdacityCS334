#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  if ( x < numRows && y < numCols )
  {
    int index = x + numRows * y;
    uchar4 rgbcolor = rgbaImage[index];
    unsigned char grey = (unsigned char)( 0.299f * rgbcolor.x + 0.587f * rgbcolor.y + 0.114f * rgbcolor.z );
    greyImage[index] = grey;
  }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  //my 1080 information
  //Maximum number of threads per multiprocessor:  2048
  //Maximum number of threads per block:           1024
  //Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
  //Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)

  int block_x = 512, block_y = 512, block_z = 32;
  const dim3 blockSize(block_x, block_y, block_z);  //TODO
  int grid_x = numRows/block_x + 1 , grid_y = numCols/block_y + 1;
  const dim3 gridSize( grid_x, grid_y, 2);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize();
  hipGetLastError();
  //checkCudaErrors(hipGetLastError());
}
